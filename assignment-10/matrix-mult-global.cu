
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int SIZE)
{
  for(int i = 0; i < SIZE; ++i)
  {
    a[i] = num;
  }
}

__global__
void matrixMultiply(float *result, float *a, float *b, int N, int SIZE)
{
  int start = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for(int i = start; i < SIZE; i += stride)
  {
    int row = i / N;

    float sum = 0;

    for (int j = 0; j < N; j++)
    {
      sum += a[row * N + j] * b[N * j + row];
    }

    result[i] = sum;
  }
}

void checkElementsAre(float target, float *array, int SIZE)
{
  for(int i = 0; i < SIZE; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values multiplied correctly.\n");
}

int main()
{
  const int N = 1024;
  const int SIZE = N * N; // sqaure matrix
  size_t size = SIZE * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, SIZE);
  initWith(4, b, SIZE);
  initWith(0, c, SIZE);

  matrixMultiply<<<100, 1024>>>(c, a, b, N, SIZE);
  hipDeviceSynchronize();

  checkElementsAre(12288, c, SIZE);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
