
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
    for (int i = 0; i < N; ++i)
    {
        a[i] = num;
    }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = start; i < N; i += stride)
    {
        result[i] = a[i] + b[i];
    }
}

void checkElementsAre(float target, float *array, int N)
{
    for (int i = 0; i < N; i++)
    {
        if (array[i] != target)
        {
            printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
            exit(1);
        }
    }
    printf("SUCCESS! All values added correctly.\n");
}

int main()
{
    const int N = 2 << 20;
    size_t size = N * sizeof(float);

    float *a;
    float *b;
    float *c;

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    initWith(3, a, N);
    initWith(4, b, N);
    initWith(0, c, N);

    addVectorsInto<<<100, 1024>>>(c, a, b, N);
    hipDeviceSynchronize();

    checkElementsAre(7, c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);
}